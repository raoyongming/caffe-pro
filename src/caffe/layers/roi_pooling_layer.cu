#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------

#include <cfloat>

#include "caffe/fast_rcnn_layers.hpp"
#include "caffe/util/gpu_util.cuh"

using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
__global__ void ROIPoolForward(
  const int nthreads, 
  const Dtype* bottom_data,
  const Dtype spatial_scale, 
  const int channels, 
  const int height,
  const int width, 
  const int pooled_height, 
  const int pooled_width,
  const Dtype* bottom_rois, 
  Dtype* top_data, 
  int* argmax_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = bottom_rois[0];
    Dtype roi_start_w = bottom_rois[1] * spatial_scale;
    Dtype roi_start_h = bottom_rois[2] * spatial_scale;
    Dtype roi_end_w = bottom_rois[3] * spatial_scale;
    Dtype roi_end_h = bottom_rois[4] * spatial_scale;

    // Force malformed ROIs to be 1x1
    Dtype roi_width = roi_end_w - roi_start_w;
    Dtype roi_height = roi_end_h - roi_start_h;

      // Compute w and h at bottom
    Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      int x1, x2, y1, y2;
      float px, py, pxmax, pymax, pxmin, pymin;
      pxmax = min(max(roi_start_w + static_cast<Dtype>(pw + 0.75) * bin_size_w, 0.001), width - 1.001);
      pymax = min(max(roi_start_h + static_cast<Dtype>(ph + 0.75) * bin_size_h, 0.001), height - 1.001);
      pxmin = min(max(roi_start_w + static_cast<Dtype>(pw + 0.25) * bin_size_w, 0.001), width - 1.001);
      pymin = min(max(roi_start_h + static_cast<Dtype>(ph + 0.25) * bin_size_h, 0.001), height - 1.001);

      Dtype out_sum = 0;

      bottom_data += (roi_batch_ind * channels + c) * height * width;

      px = pxmin;
      py = pymin;

      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);

      out_sum += (px-x1)*(py-y1) * bottom_data[int(y2*width + x2)];
      out_sum += (px-x1)*(y2-py) * bottom_data[int(y1*width + x2)];
      out_sum += (x2-px)*(py-y1) * bottom_data[int(y2*width + x1)];
      out_sum += (x2-px)*(y2-py) * bottom_data[int(y1*width + x1)];

      px = pxmax;
      py = pymax;

      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);

      out_sum += (px-x1)*(py-y1) * bottom_data[int(y2*width + x2)];
      out_sum += (px-x1)*(y2-py) * bottom_data[int(y1*width + x2)];
      out_sum += (x2-px)*(py-y1) * bottom_data[int(y2*width + x1)];
      out_sum += (x2-px)*(y2-py) * bottom_data[int(y1*width + x1)];

      px = pxmin;
      py = pymax;

      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);

      out_sum += (px-x1)*(py-y1) * bottom_data[int(y2*width + x2)];
      out_sum += (px-x1)*(y2-py) * bottom_data[int(y1*width + x2)];
      out_sum += (x2-px)*(py-y1) * bottom_data[int(y2*width + x1)];
      out_sum += (x2-px)*(y2-py) * bottom_data[int(y1*width + x1)];

      px = pxmax;
      py = pymin;

      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);

      out_sum += (px-x1)*(py-y1) * bottom_data[int(y2*width + x2)];
      out_sum += (px-x1)*(y2-py) * bottom_data[int(y1*width + x2)];
      out_sum += (x2-px)*(py-y1) * bottom_data[int(y2*width + x1)];
      out_sum += (x2-px)*(y2-py) * bottom_data[int(y1*width + x1)];
    top_data[index] = out_sum/4;
  }
}

template <typename Dtype>
void ROIPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int* argmax_data = max_idx_.mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ROIPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, spatial_scale_, channels_, height_, width_,
      pooled_height_, pooled_width_, bottom_rois, top_data, argmax_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ROIPoolBackward(
  const int nthreads, 
  const Dtype* top_diff,
  const int* argmax_data, 
  const int num_rois, 
  const Dtype spatial_scale,
  const int channels, 
  const int height, 
  const int width,
  const int pooled_height, 
  const int pooled_width, 
  Dtype* bottom_diff,
  const Dtype* bottom_rois) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = bottom_rois[0];
    Dtype roi_start_w = bottom_rois[1] * spatial_scale;
    Dtype roi_start_h = bottom_rois[2] * spatial_scale;
    Dtype roi_end_w = bottom_rois[3] * spatial_scale;
    Dtype roi_end_h = bottom_rois[4] * spatial_scale;

    Dtype roi_width = roi_end_w - roi_start_w;
    Dtype roi_height = roi_end_h - roi_start_h;

    // Compute w and h at bottom
    Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

     int x1, x2, y1, y2 ;
      float pxmin, pymin, pxmax, pymax, py, px;
      pxmax = min(max(roi_start_w + static_cast<Dtype>(pw + 0.75) * bin_size_w, 0.001), width - 1.001);
      pymax = min(max(roi_start_h + static_cast<Dtype>(ph + 0.75) * bin_size_h, 0.001), height - 1.001);
      pxmin = min(max(roi_start_w + static_cast<Dtype>(pw + 0.25) * bin_size_w, 0.001), width - 1.001);
      pymin = min(max(roi_start_h + static_cast<Dtype>(ph + 0.25) * bin_size_h, 0.001), height - 1.001);

      Dtype* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;
      Dtype diff_val = 0;
      diff_val = top_diff[index]/4;

      px = pxmin;
      py = pymin;
      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);
      caffe_gpu_atomic_add(diff_val * (px-x1)*(py-y1), offset_bottom_diff + int(y2*width + x2));
      caffe_gpu_atomic_add(diff_val * (px-x1)*(y2-py), offset_bottom_diff + int(y1*width + x2));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(py-y1), offset_bottom_diff + int(y2*width + x1));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(y2-py), offset_bottom_diff + int(y1*width + x1));

      px = pxmax;
      py = pymax;
      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);
      caffe_gpu_atomic_add(diff_val * (px-x1)*(py-y1), offset_bottom_diff + int(y2*width + x2));
      caffe_gpu_atomic_add(diff_val * (px-x1)*(y2-py), offset_bottom_diff + int(y1*width + x2));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(py-y1), offset_bottom_diff + int(y2*width + x1));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(y2-py), offset_bottom_diff + int(y1*width + x1));

      px = pxmin;
      py = pymax;
      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);
      caffe_gpu_atomic_add(diff_val * (px-x1)*(py-y1), offset_bottom_diff + int(y2*width + x2));
      caffe_gpu_atomic_add(diff_val * (px-x1)*(y2-py), offset_bottom_diff + int(y1*width + x2));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(py-y1), offset_bottom_diff + int(y2*width + x1));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(y2-py), offset_bottom_diff + int(y1*width + x1));

      px = pxmax;
      py = pymin;
      x1 = floor(px);
      x2 = ceil(px);
      y1 = floor(py);
      y2 = ceil(py);
      caffe_gpu_atomic_add(diff_val * (px-x1)*(py-y1), offset_bottom_diff + int(y2*width + x2));
      caffe_gpu_atomic_add(diff_val * (px-x1)*(y2-py), offset_bottom_diff + int(y1*width + x2));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(py-y1), offset_bottom_diff + int(y2*width + x1));
      caffe_gpu_atomic_add(diff_val * (x2-px)*(y2-py), offset_bottom_diff + int(y1*width + x1));
  }
}

template <typename Dtype>
void ROIPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int bottom_count = bottom[0]->count();
  caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
  caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);
  const int count = top[0]->count();
  const int* argmax_data = max_idx_.gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ROIPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, argmax_data, top[0]->num(), spatial_scale_, channels_,
      height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(ROIPoolingLayer);

}  // namespace caffe
